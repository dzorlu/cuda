#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Grean and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "utils.h"

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  //BlockDim is (32, 32)
  int y = threadIdx.y + blockIdx.y* blockDim.y;
  int x = threadIdx.x + blockIdx.x* blockDim.x;

  int index = numRows*y +x;
  uchar4 color = rgbaImage(index);
  float channelSum = .299f * color.x + .587f * color.y + .114f * color.z;
  greyImage[index] = channelSum;

}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  int BlockWidth = 32;
  // threads per block
  const dims blockSize(BlockWidth,BlockWidth,1);
  // block
  int blocksX = numRows/blockWidth + 1;
  int blocksY = numCols/blockWidth + 1;
  const gridSzie(blocksX, blocksY, 1 )

  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
